#include "hip/hip_runtime.h"
#include <windows.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <string.h>

int NPoints = 90000 ;
int LDimension = 3;
int Mcolumn = 90;
int HMcolumn = Mcolumn;

int TCycle = (int)(1e3);
int PTCycle = (int)(1e2);
float Rc1 = 1.0;
float LLL = (float)(TCycle * 1.0);

int Region = 2 ;
char fname_cx[100] =  "C:\\Users\\Safanajim\\Dropbox\\mFSPEx";//3.txt"  ;
//--------------------------------------------------------------------
__host__ void storeX(float *x, int N, int LDimension)
{/*  	
	FILE *fp1; 
 fp1 = fopen("H:\\Reports\\Step_by_Step_MDS\\MDS4.txt", "w");	 
	for (int i=0; i<N*LDimension; i++)
       fprintf(fp1,"%f,",x[i]);        
  fclose(fp1);
 */
 FILE *fp1;   	  
	char fname[200]="";
	strcat(fname, fname_cx);	
	char Region_str[3] = "";
	itoa(Region, Region_str, 10);
	strcat(fname, Region_str);
	strcat(fname, ".txt");
	//printf("xxxx) %s\n", fname); getchar();
 fp1 = fopen(fname, "w");	 
 int j=0;
	for (int i=0; i<N; i++)
			for (int k=0; k<LDimension; k++)
				{fprintf(fp1,"%f,",x[j]);  				
				 //printf("%d) %f\n", j, x[j]);
				 j++;				 
				}
 fclose(fp1); 
 
}
//---------------------------------------------------------------------
//----------------------------------------------------------------------------------
__host__ void SPE_CPU(int a, float *OData, float lambda,float *x, float rcut, int Dimension, int LDimension, int N)
{   //int b = blockIdx.x * blockDim.x + threadIdx.x;
				//if (b < N)   
				for(int b=0; b<N; b++)
				{			float sum = 0.0;
					   for(int k=0; k<LDimension; k++)
					      sum += (float)((x[a+ N*k] - x[b+ N*k]) * (x[a+N*k] - x[b+N*k]));												  
					   float dab = sqrt(sum);
					   float rab = dab;
			    															              															
					   sum = 0.0;															
					   for(int k=0; k<Dimension; k++)
								   sum += (float)((OData[a + N*k] - OData[b+ N*k]) * (OData[a+ N*k] - OData[b+ N*k]));	
					   rab = sqrt(sum);
							
			 	   if (((rab <= rcut) || ((rab>rcut) && (dab < rab))))
			     //if ((dab<=rcut) || ((dab > rcut)&&(dab<rab)))																	
								   {   float T;
												   T = (float)(lambda * (rab - dab) / (dab + 1e-8));																																		          								
												   for (int k=0; k<LDimension; k++)
                   x[b+N*k] += (float)(T * (x[b+N*k] - x[a+N*k]));    																																																																													
									   }													  
       } 
       //__syncthreads();
  } 
//----------------------------------------------------------------------------------
__host__ void initial_X(float *x, int N, float rcut, int LDimension)
{   int q = RAND_MAX;
    for (int i = 0; i < N*LDimension; i++) 
       x[i] = (float)(1.0 * rand() / q); 
}
//--------------------------------------------------------------------------------
__host__ void stress(float *x, float *OData, int N, float &S, float rcut, int Dimension, int LDimension)
{		int i, a, b;		
		float b1=0, b2=0;
		for (i=0; i<N; i++)
		//for (j=i+1; j<N; j++)
		{ 
			a = (int)(rand() * (float)(N-1) / (RAND_MAX+1.0));
			while(1) {
					b = (int)(rand() * (float)(N-1) / (RAND_MAX+1.0));
					if (b == a) continue;
					else break;
		}
		//a = i;
		//b = j;      
		float sum = 0.0;
        for(int k=0; k<LDimension; k++)
			sum += (float)((x[a+N*k] - x[b+N*k]) * (x[a+N*k] - x[b+N*k]));
        float dab = sqrt(sum);
								
		int k;
		sum = 0.0;
		for(k=0; k<Dimension; k++)
								sum += (float)((OData[a + N*k] - OData[b+ N*k]) * (OData[a+ N*k] - OData[b+ N*k]));	
		float rab = sqrt(sum); 
		//(abs(rab) > 0)&&
		if (((rab <= rcut) || ((rab>rcut)&&(dab < rab))))
		{  b1 += (float)((dab - rab) * (dab - rab)/(1e-8 + rab));
		   b2 += rab ;}
		}
		S = (float) (b1/(1e-8+b2));
}
//---------------------------------------------------------------------------------
__host__ void get_MD(float *OData, int N, int Dimension)
{
  int i, j;	
	float x1[90000][100]; 
 int ik;
	for (ik=0; ik<Dimension; ik++)
		{  char fname[100] =  "H:\\head\\Moffet_test"; //3\\Gulf";		
		   char Region_str[3] = "";
	    itoa(Region, Region_str, 10);
		   strcat(fname, Region_str);
		   strcat(fname, "\\Gulf");
		   
		   //char fname[100] =  "H:\\head\\Moffet_test0\\Gulf";
		
		
     int fno2= ik+1;   			
		   char buffer[3]="";
		   itoa(fno2, buffer, 10);
		   strcat(buffer,".txt");
		   strcat(fname,buffer);
		   if (ik==0)
			     printf("%3d) %s\n", ik, fname);
		   FILE *fp1;
		   float Value;
		   fp1 = fopen(fname,"r");
		   for(i=0; i<N; i++)
			   {  fscanf(fp1,"%f,", &Value);
			  	   x1[i][ik]= Value;
			   }
		   fclose(fp1);
			}
   j=0;
   for (int ik=0; ik<Dimension; ik++)
		 for (int i=0; i<N; i++)
			 { OData[j] = x1[i][ik];
					 j++;
			 }	
		 //printf("Reading originl space is done\n");			
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%                      
int main() 
{   
				int Dimension = Mcolumn; 
    int i;
    int N = NPoints;
    int k = 1;
    int ncycle = k * TCycle;
    int PrintN = PTCycle;      
    float rcut= Rc1 , lambda0 = 1.0;	   
    	
    srand ( (unsigned int)time(NULL) );
    
				float *OData, *dOData;
    OData = (float*) malloc(Dimension*N*sizeof(float));
    hipMalloc(&dOData, Dimension*N* sizeof(float));
    
    get_MD(OData, N, Dimension);
    hipMemcpy(dOData, OData, Dimension*N*sizeof(float), hipMemcpyHostToDevice);   	
    printf("---- read Md Matrix  was done ----\n");		 
				//########################################################################################
				
    float *x, *xd;
				x=(float*)malloc(N*LDimension*sizeof(float)); 
				hipMalloc(&xd,N*LDimension*sizeof(float));	   
									
				initial_X(x, N, rcut, LDimension);    
				hipMemcpy(xd,x, LDimension*N*sizeof(float),hipMemcpyHostToDevice);				

				int ThreadsPerBlock = 512;
				int BlocksNeeded = (N+ThreadsPerBlock -1)/ThreadsPerBlock ;
				dim3 dimGrid( BlocksNeeded );
				dim3 dimBlock( ThreadsPerBlock );
				/*
    printf("---------------- FSPE Starts----------------\n");
    clock_t Start0 = clock();
    //diff=(clock()-start)/(double) CLOCKS_PER_SEC
    printf("Starting time is %f\n", (float)Start0/CLOCKS_PER_SEC);
    */
				
				float S1 = LLL; 			
				float Rc_LL;				
				for(i=0; i<ncycle; i++)
					{ //int a = i;					
							int a = rand()% N;		
							float lambda = lambda0 - lambda0 * i / ncycle; 			
							//float lambda = lambda0/ (1.0+i); 	
							float LL = LLL/(1+i);			
							//float LL = LLL - LLL * i / ncycle; 				            						
							Rc_LL = LL ;
						 FSPE_GPU<<<dimGrid, dimBlock>>>(a, dOData, lambda, xd, Rc_LL, Dimension, LDimension, N);  	
						 //SPE_CPU(a, OData, lambda, x, Rc_LL, Dimension, LDimension, N);  							 						 
							if ((i% PrintN ==0) && (i !=0))
								{  hipMemcpy(x, xd, LDimension*N*sizeof(float), hipMemcpyDeviceToHost);	
								   S1 = 0.0;           
											stress(x, OData, N, S1, Rc_LL, Dimension, LDimension);
											printf("%d stress=%f\n", i, S1);
								}
			}	 
		/*	
		clock_t Start2 = clock();		
  printf("Ending  time is %f\n", (float)Start2/CLOCKS_PER_SEC);
  float diff = ((float)Start2-(float)Start0);///1000000;
  float seconds = diff /  CLOCKS_PER_SEC;
  printf("------------- Diff. time is %f\n", seconds);
  */
  
		hipMemcpy(x, xd, LDimension*N*sizeof(float), hipMemcpyDeviceToHost);	   
		storeX(x, N, LDimension);
		printf("----------------  Program End  -----------------\n");
		free(OData);
		free(x);		
		hipFree(dOData);
		hipFree(xd);
		
  //getchar();	  
		return 0;
}